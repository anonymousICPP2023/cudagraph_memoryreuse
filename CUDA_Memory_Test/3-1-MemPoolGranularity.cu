﻿/*This code tests: The granularity of the memory pool is 32MB.
Even if you use the cudaMallocAsync interface to apply for space less than 32MB,
you will directly apply for 32MB of physical memory.
Allocate less than 32MB of space at second time, and the virtual addresses are contiguous. 
It should be noted that if 16MB is allocated for the first time and 17MB is allocated for the second time, 
the address of the second time will start from start+16MB, and then 32MB of physical memory will be allocated,
that is, the allocated address will not be remapped.。*/
#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};

hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}


hipError_t test1() {
    std::cout << std::endl << "This code tests:" << "  memory pool has a granularity of 32MB and that the allocated virtual addresses are consecutive" << std::endl << std::endl;
    hipError_t cudaStatus;
    int device = 0;
    struct usageStatistics statistics = {0,0,0,0};
    hipMemPoolProps poolProps = { };//create explicit pool
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;//create stream
    cudaStatus=hipMemPoolCreate(&memPool, &poolProps);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }

    int* d_a = NULL; 
    cudaStatus = hipMallocAsync((void**)&d_a,8*1024*1024,stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "------------after mallocasync 8MB---------------" << std::endl;
    std::cout << "address of d_a is " << d_a << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    int* d_b = NULL;
    cudaStatus = hipMallocAsync((void**)&d_b, 32 * 1024 * 1024, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "------------after mallocasync 32MB---------------" << std::endl;
    std::cout << "address of d_b is " << d_b << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    int* d_c = NULL;
    cudaStatus = hipMallocAsync((void**)&d_c, 32 * 1024 * 1024, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "------------after mallocasync 32MB---------------" << std::endl;
    std::cout << "address of d_b is " << d_c << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a,stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_b, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_c, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;

}



int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test1();//Verify that the memory pool has a granularity of 32MB and that the allocated virtual addresses are consecutive
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


