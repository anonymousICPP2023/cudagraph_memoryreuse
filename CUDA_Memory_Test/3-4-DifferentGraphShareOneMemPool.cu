﻿/*This code tests: All graphs share one memory pool*/

#include <hip/hip_runtime.h>

#include <iostream>


#define SIZE 32*1024*1024

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};

hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

hipError_t test() {
    int device = 0;
    hipError_t cudaStatus;
    hipGraphExec_t graphExec1,graphExec2,graphExec3;
    hipGraph_t graph;  
    struct usageStatistics u = { 0,0,0,0 };
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream, stream1,stream2, stream3;
    int* d_a = NULL;
    int* d_b = NULL;
    int* d_c = NULL;

    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphInstantiate(&graphExec1, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_b, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphInstantiate(&graphExec2, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_c, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphInstantiate(&graphExec3, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    std::cout << std::endl << "------before launch-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    std::cout << "d_b is " << d_b << std::endl;
    std::cout << "d_c is " << d_c << std::endl;

    cudaStatus = hipGraphLaunch(graphExec1, stream1);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = GraphPoolAttrGet(device, &u);//Query graph memory pool usage 
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphLaunch(graphExec2, stream2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = GraphPoolAttrGet(device, &u);//Query graph memory pool usage 
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphLaunch(graphExec3, stream3);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = GraphPoolAttrGet(device, &u);//Query graph memory pool usage 
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    std::cout << std::endl<< "------after launch-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    std::cout << "d_b is " << d_b << std::endl;
    std::cout << "d_c is " << d_c << std::endl;



    return hipSuccess;
}



int main(int argc, char** argv)
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

