#include "hip/hip_runtime.h"
﻿/*This code tests:The effect of performance improvement of in-stream reuse*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>


#define LOOPTIMES 20  //loop times
#define SIZE 64*1024*1024  //size of allocation



__global__ void clockBlock(clock_t clock_count) { //kernel
    unsigned int start_clock = (unsigned int)clock();

    clock_t clock_offset = 0;

    while (clock_offset < clock_count) {
        unsigned int end_clock = (unsigned int)clock();
        clock_offset = (clock_t)(end_clock - start_clock);
    }
}

hipError_t test1() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMallocAsync+hipFreeAsync---No synchronize in each loop" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);



    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock <<<1, 1, 0, stream >>> (time_clocks);
        cudaStatus = hipFreeAsync(d_a, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test1 time is %f\n", time);
    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test2() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMallocAsync+hipFreeAsync---Synchronize in each loop" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);



    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock << <1, 1, 0, stream >> > (time_clocks);
        cudaStatus = hipFreeAsync(d_a, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipStreamSynchronize(stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamSynchronize failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test2 time is %f\n", time);

    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test3() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMallocAsync---Memory is only allocated but not released" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* device[LOOPTIMES];
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);



    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&device[i], SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock <<<1, 1, 0, stream >>> (time_clocks);
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }

    printf("test3 time is %f\n", time);

    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipFreeAsync(device[i], stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test4() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMallocAsync+hipFreeAsync---Set threshold max and synchronize" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    hipMemPool_t memPool;
    cudaStatus = hipDeviceGetDefaultMemPool(&memPool, 0); 
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetDefaultMemPool failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);
    long unsigned int setVal = UINT64_MAX;
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock << <1, 1, 0, stream >> > (time_clocks);
        cudaStatus = hipFreeAsync(d_a, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipStreamSynchronize(stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamSynchronize failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test4 time is %f\n", time);
    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test5() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMallocAsync+hipFreeAsync---Set threshold 32MB and synchronize" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    hipMemPool_t memPool;
    cudaStatus = hipDeviceGetDefaultMemPool(&memPool, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetDefaultMemPool failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);
    long unsigned int setVal = 32*1024*1024;
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock << <1, 1, 0, stream >> > (time_clocks);
        cudaStatus = hipFreeAsync(d_a, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipStreamSynchronize(stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamSynchronize failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test5 time is %f\n", time);
    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test6() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMalloc+hipFree" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* d_a = NULL;
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);



    cudaStatus = hipEventRecord(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMalloc((void**)&d_a, SIZE);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock << <1, 1, 0 >> > (time_clocks);
        cudaStatus = hipFree(d_a);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test6 time is %f\n", time);
    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test7() {
    std::cout << std::endl << "This code tests: The effect of performance improvement of in-stream reuse" << std::endl << std::endl;
    std::cout << "hipMalloc---Memory is only allocated and not freed" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* device[LOOPTIMES];
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);

    cudaStatus = hipEventRecord(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMalloc((void**)&device[i], SIZE);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        clockBlock << <1, 1, 0 >> > (time_clocks);
    }

    cudaStatus = hipEventRecord(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test7 time is %f\n", time);

    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipFree(device[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
    }
        cudaStatus = hipEventDestroy(start);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventDestroy failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventDestroy(stop);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventDestroy failed!");
            return hipErrorInvalidValue;
        }
        return hipSuccess;
    
}
int main(int argc, char** argv)
{

    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

  //  cudaStatus = test1();//hipMallocAsync+hipFreeAsync---Do not synchronize in each loop---Memory is reused
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    } 
  //  cudaStatus = test2();//hipMallocAsync+hipFreeAsync---Synchronize in each loop---Memory is not reused
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test2 failed!");
        return 1;
    }
   // cudaStatus = test3();//hipMallocAsync---Memory is only allocated but not released---Memory is not reused
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test3 failed!");
        return 1;
    }
    //cudaStatus = test4();//hipMallocAsync+hipFreeAsync---Set threshold max and synchronize---Memory is reused
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test4 failed!");
        return 1;
    }
   // cudaStatus = test5();//hipMallocAsync+cudaFreeAsyncc---Set threshold 32MB and synchronize---Partial memory is reused
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test5 failed!");
        return 1;
    }
    //cudaStatus = test6();//hipMalloc+hipFree
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test6 failed!");
        return 1;
    }
    cudaStatus = test7();//hipMalloc---Memory is only allocated and not freed
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test7 failed!");
        return 1;
    }

        // hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

