﻿/*This code tests: compare allocA->freeA->allocB with allocA->allocB->freeA*/
#include <hip/hip_runtime.h>

#include <iostream>


#define SIZE 416*1024*1024


struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};
hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}


hipError_t test1() {
    hipError_t cudaStatus;
    int* d_a = NULL,*d_b = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraphExec_t graphExec;
    struct usageStatistics statistics = { 0,0,0,0 };


    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_b, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("allocA->freeA->allocB is %f\n", time);
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test2() {
    hipError_t cudaStatus;
    int* d_a = NULL, * d_b = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraphExec_t graphExec;
    struct usageStatistics statistics = { 0,0,0,0 };


    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_b, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("allocA->allocB->freeA is %f\n", time);
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
int main(int argc, char** argv)
{

    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    std::cout << std::endl << "This code tests:" << " compare allocA->freeA->allocB with allocA->allocB->freeA" << std::endl << std::endl;
    cudaStatus = test1();//allocA->freeA->allocB
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }
    cudaStatus = test2();//allocA->allocB->freeA
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test2 failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;

}

