﻿/*This code tests:Graph memory pool does not release memory after synchronization*/
#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};


hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

hipError_t test() {
    std::cout << std::endl << "This code tests:Graph memory pool does not release memory after synchronization" << std::endl << std::endl;
    hipError_t cudaStatus;
    struct usageStatistics statistics= { 0,0,0,0 };
    hipGraphExec_t graphExec;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph,0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream;
    int* d_a = NULL;


    int SIZE = 1 * 1024 * 1024;

    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    std::cout << std::endl << "------before instantiate-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);//instantiate graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }


    std::cout << std::endl << "------before launch-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }

    std::cout << std::endl << "------after launch-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamSynchronize(stream);//stream synchronization
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }


    std::cout << std::endl << "------after synchronize-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGraphMemTrim(0);//TrimTo
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGraphMemTrim failed!");
        return hipErrorInvalidValue;
    }
    std::cout << std::endl << "------after trimto-------" << std::endl;
    std::cout << "d_a is " << d_a << std::endl;
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }
    cudaStatus = test();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test failed!");
        return 1;
    }
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

