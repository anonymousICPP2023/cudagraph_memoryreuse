﻿/*This code tests:Memory reuse of nodes in the graph 
Can the original node 1G be reused with two subsequent 0.5G nodes?*/
#include <hip/hip_runtime.h>

#include <iostream>



struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};
hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

void prepareAllocParams(hipMemAllocNodeParams* allocParams, size_t bytes,
    int device) {
    memset(allocParams, 0, sizeof(*allocParams));

    allocParams->bytesize = bytes;
    allocParams->poolProps.allocType = hipMemAllocationTypePinned;
    allocParams->poolProps.location.id = device;
    allocParams->poolProps.location.type = hipMemLocationTypeDevice;
}

hipError_t test1() {
    std::cout << std::endl << "This code tests: Can the node be reused with two subsequent nodes? " << std::endl << std::endl;
    hipError_t cudaStatus;
    float* d_a = NULL,* d_b = NULL,* d_c = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph,0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraphExec_t graphExec;
    struct usageStatistics statistics = { 0 };
    hipMemAllocNodeParams allocParamsA, allocParamsB, allocParamsC;
    hipGraphNode_t allocNodeA, allocNodeB, allocNodeC, freeNodeA;
    long long int size1 = 1024 * 1024 * 1024, size2 = 608 * 1024 * 1024, size3 = 416 * 1024 * 1024;
    std::cout << "allocA size is " << size1 << std::endl;
    std::cout << "allocB size is " << size2 << std::endl;
    std::cout << "allocC size is " << size3 << std::endl << std::endl;
    prepareAllocParams(&allocParamsA, 512 * 1024 * 1024, 0);
    prepareAllocParams(&allocParamsB, 608 * 1024 * 1024, 0);
    prepareAllocParams(&allocParamsC, 416 * 1024 * 1024, 0);

    cudaStatus = hipGraphAddMemAllocNode(&allocNodeA, graph, NULL, 0, &allocParamsA);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphAddMemAllocNode failed!");
        return hipErrorInvalidValue;
    }
    d_a = (float*)allocParamsA.dptr;
    cudaStatus = hipGraphAddMemFreeNode(&freeNodeA, graph, &allocNodeA, 1, (void*)d_a);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphAddMemFreeNode failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphAddMemAllocNode(&allocNodeB, graph, &freeNodeA, 1, &allocParamsB);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphAddMemAllocNode failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphAddMemAllocNode(&allocNodeC, graph, &freeNodeA, 1, &allocParamsC);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphAddMemAllocNode failed!");
        return hipErrorInvalidValue;
    }
    d_b = (float*)allocParamsB.dptr;
    d_c = (float*)allocParamsC.dptr;

    std::cout << "allocA address is " << d_a << std::endl;
    std::cout << "allocB address is " << d_b << std::endl;
    std::cout << "allocC address is " << d_c << std::endl << std::endl;
    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = GraphPoolAttrGet(0, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}


int main(int argc, char** argv)
{

    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test1();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

