﻿/*This code tests: when Alloc->Free->New alloc    
If the size of new alloc is larger than alloc, can memory be reused
If the size of new alloc is equal to alloc, can memory be reused
If the size of new alloc is smaller to alloc, can memory be reused
*/

#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};
//Create a graph with only malloc node with stream capture
hipError_t createGraphWithStreamCapture(hipGraphExec_t* graphExec) {
    hipError_t cudaStatus;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream;
    int* d_a = NULL;

    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMallocAsync((void**)&d_a, 1 << 30, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphInstantiate(graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }

    return hipSuccess;
}


hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}


hipError_t test() {
    std::cout << std::endl << "This code tests:" << "If the size of new alloc is larger than alloc, can memory be reused" << std::endl << std::endl;
    hipError_t cudaStatus;
    hipGraphExec_t graphExec;
    hipGraph_t graph;
    struct usageStatistics u = { 0,0,0,0 };
    cudaStatus = hipGraphCreate(&graph, 0);
    hipStream_t stream;
    int* d_a = NULL;
    int* d_b = NULL;

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = createGraphWithStreamCapture(&graphExec);//stream capture creates executable graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "createGraphWithStreamCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "createGraphWithStreamCapture failed!");
        return hipErrorInvalidValue;
    }

    long long int size1 = 32 * 1024 * 1024;
    long long int size2 = 64 * 1024 * 1024;

    std::cout << "alloc size is     " << size1 << std::endl ;
    std::cout << "new alloc size is " << size2 << std::endl << std::endl;

    cudaStatus = hipMallocAsync((void**)&d_a, size1, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_b, size2, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "alloc address is     " << d_a << std::endl ;
    std::cout << "new alloc address is " << d_b << std::endl << std::endl;
    cudaStatus = GraphPoolAttrGet(0, &u);//query graph memory pool usage
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GraphPoolAttrGet failed!");
        return hipErrorInvalidValue;
    };
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return cudaStatus;



}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
