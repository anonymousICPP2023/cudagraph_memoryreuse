#include "hip/hip_runtime.h"
﻿/*This code tests:An executable graph is launched into multiple streams, even if there is no malloc and free nodes, it can only be serialized;
a graph which has no alloc nodes can be instantiated to multiple executable graphs; 
a graph which has alloc nodes cannot be instantiated to multiple executable graphs.
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define LOOPTIMES 10
#define SIZE 32*1024*1024

__global__ void clockBlock(clock_t clock_count) {
    unsigned int start_clock = (unsigned int)clock();

    clock_t clock_offset = 0;

    while (clock_offset < clock_count) {
        unsigned int end_clock = (unsigned int)clock();
        clock_offset = (clock_t)(end_clock - start_clock);
    }
}
void prepareAllocParams(hipMemAllocNodeParams* allocParams, size_t bytes,
    int device) {
    memset(allocParams, 0, sizeof(*allocParams));

    allocParams->bytesize = bytes;
    allocParams->poolProps.allocType = hipMemAllocationTypePinned;
    allocParams->poolProps.location.id = device;
    allocParams->poolProps.location.type = hipMemLocationTypeDevice;
}

hipError_t createGraphWithMalloc(hipGraphExec_t* graphExec) {
    hipError_t cudaStatus;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream;
    int* d_a = NULL;
    float kernelTime = 5000;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    clockBlock <<<1, 1, 0, stream >>> (time_clocks);
    cudaStatus = hipFreeAsync(d_a,stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}

hipError_t createGraphWithoutMalloc(hipGraphExec_t* graphExec) {
    hipError_t cudaStatus;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream;
    float kernelTime = 5000;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }

    clockBlock << <1, 1, 0, stream >> > (time_clocks);

    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}

hipError_t test1() {
    hipError_t cudaStatus;
    hipGraphExec_t graphExec;
    hipStream_t stream[LOOPTIMES];    
    cudaStatus = createGraphWithMalloc(&graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipStreamCreateWithFlags(&(stream[i]), hipStreamNonBlocking);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphDestroy failed!");
            return hipErrorInvalidValue;
        }
    }
        
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipGraphLaunch(graphExec, stream[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphDestroy failed!");
            return hipErrorInvalidValue;
        }
    }
    return cudaStatus;
}
hipError_t test2() {
    hipError_t cudaStatus;
    hipGraphExec_t graphExec;
    hipStream_t stream[LOOPTIMES];
    cudaStatus = createGraphWithoutMalloc(&graphExec);
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipStreamCreateWithFlags(&(stream[i]), hipStreamNonBlocking);
        if (cudaStatus != hipSuccess) {
             fprintf(stderr, "hipGraphDestroy failed!");
             return hipErrorInvalidValue;
        }   
    }
       
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipGraphLaunch(graphExec, stream[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphDestroy failed!");
            return hipErrorInvalidValue;
        }
    }
    return cudaStatus;
}

hipError_t test3() {
    hipError_t cudaStatus;
    hipGraphExec_t graphExec[LOOPTIMES];
    hipStream_t stream[LOOPTIMES];
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipStreamCreateWithFlags(&(stream[i]), hipStreamNonBlocking);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamCreateWithFlags failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = createGraphWithMalloc(&(graphExec[i]));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "createGraphWithMalloc failed!");
            return hipErrorInvalidValue;
        }
    }

    for (int i = 0; i < LOOPTIMES; i++){
        cudaStatus = hipGraphLaunch(graphExec[i], stream[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphLaunch failed!");
            return hipErrorInvalidValue;
        }
    }
    return cudaStatus;
}

hipError_t test4() {
    hipError_t cudaStatus;
    hipGraphExec_t graphExec[LOOPTIMES];
    hipStream_t stream[LOOPTIMES];
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipStreamCreateWithFlags(&(stream[i]), hipStreamNonBlocking);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphLaunch failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = createGraphWithoutMalloc(&(graphExec[i]));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphLaunch failed!");
            return hipErrorInvalidValue;
        }
    }

    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipGraphLaunch(graphExec[i], stream[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphLaunch failed!");
            return hipErrorInvalidValue;
        }
    }
    return cudaStatus;
}

hipError_t test5() {
    hipError_t cudaStatus;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraphExec_t graphExec1, graphExec2;
    hipStream_t stream;
    float* d_a = NULL;
    hipMemAllocNodeParams allocParamsA;
    hipGraphNode_t allocNodeA,  freeNodeA;
    prepareAllocParams(&allocParamsA, 32 * 1024 * 1024, 0);
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphAddMemAllocNode(&allocNodeA, graph, NULL, 0, &allocParamsA);//create graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphAddMemAllocNode failed!");
        return hipErrorInvalidValue;
    }
    d_a = (float*)allocParamsA.dptr;
    cudaStatus = hipGraphAddMemFreeNode(&freeNodeA, graph, &allocNodeA, 1, (void*)d_a);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphAddMemFreeNode failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(&graphExec1, graph, NULL, NULL, 0);//instantiate graph first time
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "first time hipGraphInstantiate failed!");
        exit(0);
    }
    cudaStatus = hipGraphInstantiate(&graphExec2, graph, NULL, NULL, 0);//instantiate graph second time
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "second time hipGraphInstantiate failed!");
        exit(0);
    }
    return cudaStatus;

}

hipError_t test6() {
    hipError_t cudaStatus;
    hipGraphExec_t graphExec1, graphExec2;
    hipStream_t stream,stream1,stream2;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphCreate failed!");
        return hipErrorInvalidValue;
    }
    float kernelTime = 5000;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    clockBlock <<<1, 1, 0, stream >>> (time_clocks);
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipGraphInstantiate(&graphExec1, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "1 hipGraphInstantiate failed!");
        exit(0);
    }
    cudaStatus = hipGraphInstantiate(&graphExec2, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "2 hipGraphInstantiate failed!");
        exit(0);
    }

    cudaStatus = hipGraphLaunch(graphExec1, stream1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphLaunch(graphExec2, stream2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphLaunch failed!");
        return hipErrorInvalidValue;
    }
    return cudaStatus;
}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }
    //Viewing results in visual profiler
    cudaStatus =test1();//Single executable graph is launched into multiple streams Has malloc nodes
    //cudaStatus =test2();//Single executable graph is launched into multiple streams No malloc nodes
    //cudaStatus =test3();//Multiple executable graphs are launched into multiple streams Have malloc nodes
    //cudaStatus =test4();//Multiple executable graphs are launched into multiple streams No malloc nodes
    //cudaStatus = test5();//A graph with memory allocation nodes cannot instantiate multiple executable graphs
    //cudaStatus =test6();//A graph without memory allocation nodes can instantiate multiple executable graphs
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

