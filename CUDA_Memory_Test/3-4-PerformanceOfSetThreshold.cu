﻿/*This code tests:
compare performance differences of cudaMallocAsync between setting the threshold and do not set the threshold. */

#include <hip/hip_runtime.h>

#include <iostream>

#define LOOPTIMES 1000 //loop times
#define SIZE 32 * 1024 * 1024 //size of allocation

hipError_t test1() {
    int device = 0;// Choose which GPU to run on, change this on a multi-GPU system.
    std::cout << std::endl << "This code tests:  performance differences of hipMallocAsync between setting the threshold and do not set the threshold." << std::endl << std::endl;
    std::cout << "set threshold---cudaMallocAsync+cudaFreeAsync---Synchronize in each loop" << std::endl << std::endl;
    hipError_t cudaStatus ;
    int* d_a = NULL;
    hipMemPoolProps poolProps = { };//set pool properties
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);//create explicit pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }

    unsigned long long int setVal = UINT64_MAX;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipFreeAsync(d_a, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipStreamSynchronize(stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamSynchronize failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test1 time is %f\n", time);

    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
hipError_t test2() {
    std::cout << std::endl << "This code tests:  performance differences of hipMallocAsync between setting the threshold and do not set the threshold." << std::endl << std::endl;
    std::cout << "no threshold---cudaMallocAsync+cudaFreeAsync---Synchronize in each loop" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipFreeAsync(d_a, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipStreamSynchronize(stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamSynchronize failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("test2 time is %f\n", time);

    cudaStatus = hipEventDestroy(start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventDestroy(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}
int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

     cudaStatus = test1();//set threshold---cudaMallocAsync+cudaFreeAsync---Synchronize in each loop
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }
    //  cudaStatus = test2();//no threshold---cudaMallocAsync+cudaFreeAsync---Synchronize in each loop
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test2 failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

