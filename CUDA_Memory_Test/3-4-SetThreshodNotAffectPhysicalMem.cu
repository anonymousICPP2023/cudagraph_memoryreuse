﻿/*This code tests: Whether the physical memory changes when the threshold is set for the memory pool*/
#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};

hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent,&(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}


hipError_t test1() {
    std::cout << std::endl<<"This code tests:" << " Whether the physical memory changes when the threshold is set for the memory pool" << std::endl << std::endl;
    int device = 0;
    hipError_t cudaStatus;

    struct usageStatistics statistics = { 0,0,0,0 };

    hipMemPoolProps poolProps = { };//set pool properties
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;
    cudaStatus=hipMemPoolCreate(&memPool, &poolProps);//create explicit pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "before set threshold " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    unsigned int setVal = 1 << 30;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after set threshold " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return cudaStatus;
}


int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus=hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test1();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;   
}
