#include "hip/hip_runtime.h"
﻿/*This code tests: Graph memory should avoid frequent trimming*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>


#define LOOPTIMES 10
#define SIZE 1024*1024*1024


struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};
hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}


__global__ void clockBlock(clock_t clock_count) {
    unsigned int start_clock = (unsigned int)clock();

    clock_t clock_offset = 0;

    while (clock_offset < clock_count) {
        unsigned int end_clock = (unsigned int)clock();
        clock_offset = (clock_t)(end_clock - start_clock);
    }
}

hipError_t test1() {
    std::cout << "Do not trim the graph after each launch in this test" << std::endl << std::endl;
    int device = 0;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);


    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    clockBlock <<<1, 1, 0, stream >>> (time_clocks);
    cudaStatus = hipFreeAsync(d_a, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);//instantiate graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphLaunch failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }
    printf("time is %f\n\n", time);
    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}

hipError_t test2() {
    std::cout << "Trim the graph after each launch in this test" << std::endl << std::endl;
    int device = 0;
    hipError_t cudaStatus;
    int* d_a = NULL;
    hipStream_t stream;
    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    float kernelTime = 50;  // time for each thread to run in microseconds
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceProperties(&deviceProp, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        return hipErrorInvalidValue;
    }
    clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);


    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);//create graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_a, SIZE, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    clockBlock << <1, 1, 0, stream >> > (time_clocks);
    cudaStatus = hipFreeAsync(d_a, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);//instantiate graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipEventRecord(start, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    for (int i = 0; i < LOOPTIMES; i++) {
        cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphLaunch failed!");
            return hipErrorInvalidValue;
        }

        cudaStatus = hipStreamSynchronize(stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipStreamSynchronize failed!");
            return hipErrorInvalidValue;
        }

        cudaStatus = hipDeviceGraphMemTrim(0);//trim graph memory pool
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceGraphMemTrim failed!");
            return hipErrorInvalidValue;
        }

    }

    cudaStatus = hipEventRecord(stop, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventSynchronize(stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventElapsedTime(&time, start, stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventElapsedTime failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    printf("time is %f\n", time);
    return hipSuccess;
}
int main(int argc, char** argv)
{

    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    std::cout << std::endl << "This code tests: Graph memory should avoid frequent trimming" << std::endl << std::endl;

   cudaStatus = test1(); //do not trim
   cudaStatus = test2(); //trim
   if (cudaStatus != hipSuccess) {
       fprintf(stderr, "test failed!");
       return 1;
   }


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

