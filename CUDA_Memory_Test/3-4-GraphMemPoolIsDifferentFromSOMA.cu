﻿/*This code tests:When creating an allocation graph,
the allocations come from the graph memory pool instead of the default memory pool
*/

#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};

hipError_t GraphPoolAttrGet(int  device, struct usageStatistics* statistics)
{
    std::cout << "-------Graph MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}


hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

//Create a graph with only malloc node with stream capture
hipError_t createGraphWithStreamCapture(hipGraphExec_t* graphExec) {
    hipError_t cudaStatus;
    hipGraph_t graph;
    cudaStatus = hipGraphCreate(&graph, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetGraphMemAttribute failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream;
    int* d_a = NULL;

    cudaStatus = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " hipStreamCreateWithFlags failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamBeginCapture failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMallocAsync((void**)&d_a, 1 << 30, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamEndCapture(stream, &graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamEndCapture failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphInstantiate(graphExec, graph, NULL, NULL, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphInstantiate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphDestroy(graph);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphDestroy failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }

    return hipSuccess;
}

hipError_t test() {
    std::cout << std::endl << "This code tests: When creating an allocation graph," << std::endl;
    std::cout << " the allocations come from the graph memory pool instead of the default memory pool" << std::endl << std::endl;
    hipError_t cudaStatus;
    int device = 0;
    struct usageStatistics u = { 0,0,0,0 };

    hipMemPool_t memPool;

    cudaStatus = hipDeviceGetDefaultMemPool(&memPool, 0); //get default pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    hipGraphExec_t graphExec;

    cudaStatus = createGraphWithStreamCapture(&graphExec);//create executable graph with stream capture
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    hipStream_t stream;

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipGraphLaunch(graphExec, stream);//launch graph
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = poolAttrGet(memPool, &u);//Query the default pool memory usage
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = GraphPoolAttrGet(device,&u);//Query graph memory pool usage
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipGraphExecDestroy(graphExec);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGraphExecDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;


}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }
    cudaStatus = test();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
