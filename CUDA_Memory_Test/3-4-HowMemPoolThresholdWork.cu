﻿/*This code tests:
1.if the threshold is set, the physical memory will be released during each synchronization operation
2.if the threshold is not set, the physical memory will be fully released during each synchronization operation.
*/
#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};

hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}



hipError_t with_threshold() {
    hipError_t cudaStatus;
    int device = 0; // Choose which GPU to run on, change this on a multi-GPU system.
    struct usageStatistics statistics = { 0,0,0,0 };

    hipMemPoolProps poolProps = { };//create explicit pool
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;//create stream
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "before set threshold " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    unsigned int setVal =  10 * 1024 * 1024;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "after set threshold " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


    int* cn = 0ULL;

    cudaStatus = hipMallocAsync((void**)&cn, 1*1024*1024*1024, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaStreamSynchroniz failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "after mallocasync " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipFreeAsync(cn, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "after freeasync and streamsync" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipMemPoolTrimTo(memPool, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolTrimTo failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "after trim to " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;

}

hipError_t without_threshold() {
    hipError_t cudaStatus;
    int device = 0; // Choose which GPU to run on, change this on a multi-GPU system.
    struct usageStatistics statistics = { 0,0,0,0 };

    hipMemPoolProps poolProps = { };//create explicit pool
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;//create stream
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }


    int* cn = 0ULL;

    cudaStatus = hipMallocAsync((void**)&cn, 1 * 1024 * 1024 * 1024, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaStreamSynchroniz failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "after mallocasync " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipFreeAsync(cn, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }

    std::cout << "after freeasync and streamsync" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;

}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }
    std::cout << std::endl << "This code tests:memory pool defragmentation" << std::endl << std::endl;
    cudaStatus = with_threshold();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "free1_3 failed!");
        return 1;
    }


    cudaStatus = without_threshold();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "free2_3 failed!");
        return 1;
    }
    // cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
